#include <iostream>
#include <hip/hip_runtime.h>
#define vect_len 16
using namespace std;

const int blocksize = 16;

// __global__ decorator signifies a kernel that can be called from the host
__global__ void vect_add(int *a, int *b)
{
	a[threadIdx.x] += b[threadIdx.x];
}

int main(){
	const int vect_size = vect_len*sizeof(int);
	int * vect1=(int*)malloc(vect_size);
	int * vect2=(int*)malloc(vect_size);
	int * result=(int*)malloc(vect_size);
 	bool flag;
	for(int i = 0; i < vect_len; i++)
	{
		vect1[i] = i;
		vect2[i] = 2 * i;
	}
	int *ad, *bd;
	// initialize device memory
	hipMalloc( (void**)&ad, vect_size );
	hipMalloc( (void**)&bd, vect_size );
	// copy data to device
	hipMemcpy( ad, vect1, vect_size, hipMemcpyHostToDevice );
	hipMemcpy( bd, vect2, vect_size, hipMemcpyHostToDevice );
	// setup block and grid size	
	dim3 dimBlock( blocksize, 1, 1);
	dim3 dimGrid( vect_len/blocksize, 1 , 1);
	// call device kernel
	//vect_add<<<dimGrid, dimBlock>>>(ad, bd);
	vect_add<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy( result, ad, vect_size, hipMemcpyDeviceToHost );

	//Verify
	flag = true;
	for(int i = 0; i < vect_len; i++)
	{
		if(result[i] != vect1[i] + vect2[i])
		{
			cout << "Verification fail at " << i << endl;
			flag = false;
			break;
		}
	}
	if(flag)
		cout << "Verification passes." <<endl;
	// free device memory
	hipFree( ad );
	hipFree( bd );
	free(vect1);
	free(vect2);
	free(result);
	return EXIT_SUCCESS;
}


