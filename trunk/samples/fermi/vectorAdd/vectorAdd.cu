#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 3
 * of the programming guide with some additions like error checking.
 *
 */

// Includes
#include <stdio.h>
#include <cutil_inline.h>
#include <shrQATest.h>

// Variables
float* h_A;
float* h_B;
float* h_C;
float* d_A;
float* d_B;
float* d_C;
bool noprompt = false;

// Functions
void CleanupResources(void);
void RandomInit(float*, int);
void ParseArguments(int, char**);

// Device code
__global__ void VecAdd(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

// Host code
int main(int argc, char** argv)
{
    shrQAStart(argc, argv);

    printf("Vector Addition\n");
    int N = 50000;
    size_t size = N * sizeof(float);
    ParseArguments(argc, argv);

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    if (h_A == 0) CleanupResources();
    h_B = (float*)malloc(size);
    if (h_B == 0) CleanupResources();
    h_C = (float*)malloc(size);
    if (h_C == 0) CleanupResources();
    
    // Initialize input vectors
    RandomInit(h_A, N);
    RandomInit(h_B, N);

    // Allocate vectors in device memory
    cutilSafeCall( hipMalloc((void**)&d_A, size) );
    cutilSafeCall( hipMalloc((void**)&d_B, size) );
    cutilSafeCall( hipMalloc((void**)&d_C, size) );

    // Copy vectors from host memory to device memory
    cutilSafeCall( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    cutilCheckMsg("kernel launch failure");
#ifdef _DEBUG
    cutilSafeCall( cutilDeviceSynchronize() );
#endif

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    cutilSafeCall( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
    
    // Verify result
    int i;
    for (i = 0; i < N; ++i) {
        float sum = h_A[i] + h_B[i];
        if (fabs(h_C[i] - sum) > 1e-5)
            break;
    }

    CleanupResources();
    shrQAFinishExit(argc, (const char **)argv, (i==N) ? QA_PASSED : QA_FAILED);
}

void CleanupResources(void)
{
    // Free device memory
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
        
    cutilDeviceReset();
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

// Parse program arguments
void ParseArguments(int argc, char** argv)
{
    for (int i = 0; i < argc; ++i) {
        if (strcmp(argv[i], "--noprompt") == 0 ||
            strcmp(argv[i], "-noprompt") == 0) 
        {
            noprompt = true;
            break;
        }
    }
}
